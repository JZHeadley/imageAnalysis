#include "hip/hip_runtime.h"
#include "imageAnalysis.h"

#include <stdio.h>
#include <stdlib.h>
#include <iostream>

#include <omp.h>

static void CheckCudaErrorAux(const char *, unsigned, const char *, hipError_t);

#define CUDA_CHECK_RETURN(value) CheckCudaErrorAux(__FILE__,__LINE__, #value, value);

__global__ void convertRGBToGrayscaleLuminance(unsigned char *image, int width, int height, int numPixels, int channels, unsigned char *output) {
    int tid = blockDim.x * blockIdx.x + threadIdx.x;
    int row = tid / width;
    int column = tid - ((tid / width) * width);
    if ((tid < numPixels)) {
        output[row * width + column] = (unsigned char) ((.21 * image[row * width + column]) + (.72 * image[row * width + column + numPixels]) + (.07 * image[row * width + column + (2 * numPixels)]));
//        printf("%i %i\n", row, column);
    }
    return;
}

__global__ void convertRGBToGrayscaleAverage(unsigned char *image, int width, int height, int numPixels, int channels, unsigned char *output) {
    int tid = blockDim.x * blockIdx.x + threadIdx.x;
    int row = tid / width;
    int column = tid - ((tid / width) * width);
    if ((tid < numPixels)) {
        output[row * width + column] = (unsigned char) ((image[row * width + column] + image[row * width + column + numPixels] + image[row * width + column + (2 * numPixels)]) / 3);
//        printf("%i %i\n", row, column);
    }
    return;
}


void convertRGBToGrayscale(RGBImage *d_rgb, Image *d_gray, int method) {
    /*
    don't think you mentioned a grayscale conversion method so I looked it up and used this page as my guide
    https://www.johndcook.com/blog/2009/08/24/algorithms-convert-color-grayscale/
    */
    int totalPixels = d_rgb->width * d_rgb->height;
    int threadsPerBlock = 256;
    int blocksPerGrid = (totalPixels + threadsPerBlock - 1) / threadsPerBlock;
    hipError_t err;
    switch (method) {
        case 0:
            // luminance method
            CUDA_CHECK_RETURN(hipMalloc((void **) &(d_gray->image), (int) sizeof(unsigned char) * d_rgb->width * d_rgb->height));
            printf("Using the luminance method...%i %i %i %p %p\n", threadsPerBlock, blocksPerGrid, d_rgb->channels, d_gray->image, d_rgb->image);
            convertRGBToGrayscaleLuminance<< < threadsPerBlock, blocksPerGrid>> > (d_rgb->image, d_rgb->width, d_rgb->height, totalPixels, d_rgb->channels, d_gray->image);
            err = hipGetLastError();
            if (err != hipSuccess) {
                printf("Error: %s\n", hipGetErrorString(err));
            }
            d_gray->width = d_rgb->width;
            d_gray->height = d_rgb->height;
            break;
        case 1:
            // average method
            printf("Using the average method...\n");
            CUDA_CHECK_RETURN(hipMalloc((void **) &(d_gray->image), (int) sizeof(unsigned char) * d_rgb->width * d_rgb->height));

            convertRGBToGrayscaleAverage<< < threadsPerBlock, blocksPerGrid>> > (d_rgb->image, d_rgb->width, d_rgb->height, totalPixels, d_rgb->channels, d_gray->image);
            d_gray->width = d_rgb->width;
            d_gray->height = d_rgb->height;
            break;
        default:
            printf("WTF why are we defaulting?\n");
            break;
    }
}

__global__ void calcHistogram(unsigned char *data, int width, int numPixels, int *histogram) {
    int tid = blockDim.x * blockIdx.x + threadIdx.x;
    int row = tid / width;
    int column = tid - ((tid / width) * width);
    if (tid < numPixels) {
        int val = data[row * width + column];
        if (val != 0)
            atomicAdd(&histogram[val], 1);
    }
    return;
}

void calculateHistogram(Image *image, int *h_histogram, int *d_histogram) {
    int totalPixels = image->width * image->height;
    int threadsPerBlock = 512;
//    int operationsPerThread = 10;
//    int numOperations = totalPixels / operationsPerThread;
//    int blocksPerGrid = (numOperations + threadsPerBlock - 1) / threadsPerBlock;
    int blocksPerGrid = (totalPixels + threadsPerBlock - 1) / threadsPerBlock;
    CUDA_CHECK_RETURN(hipMalloc(&d_histogram, (int) sizeof(int) * 256));
    CUDA_CHECK_RETURN(hipMemset(d_histogram, 0, 256 * sizeof(int)));

    calcHistogram<< < threadsPerBlock, blocksPerGrid, 0>> > (image->image, image->width, totalPixels, d_histogram);

    CUDA_CHECK_RETURN(hipMemcpy(h_histogram, d_histogram, sizeof(int) * 256, hipMemcpyDeviceToHost));
}


void equalizeHistogram(int *original, int *mappings, int numPixels) {
//#pragma omp parallel num_threads(2)
    int numColors = 256;

    float pdf[256];
    float cdf[256];
    // tried to use openmp and speed this up more but something is weird with openmp + cuda + cmake and it only ever ran on 1 thread for me
//#pragma omp parallel for default (none) shared(numColors, original, pdf, cdf)
    for (int i = 0; i < numColors; i++) {
//        int threadId = omp_get_thread_num();
//        printf("Thread %i reporting for %i\n", omp_get_thread_num(), i);
        pdf[i] = original[i] / (float) numPixels;
        cdf[i] = pdf[i];
        if (i > 0) {
            cdf[i] = cdf[i] + cdf[i - 1];
            mappings[i] = (int) (cdf[i] * 255);
        } else {
            mappings[i] = (int) (cdf[i] * 255);
        }
    }
//#pragma omp parallel end
}

__global__ void equalizeImage(unsigned char *image, int width, int numPixels, int *mappings, unsigned char *output) {
    int tid = blockDim.x * blockIdx.x + threadIdx.x;
    int row = tid / width;
    int column = tid - ((tid / width) * width);
    if ((tid < numPixels)) {
        output[row * width + column] = (unsigned char) (mappings[image[row * width + column]]);
    }
    return;
}

void equalizeImageWithHist(Image *image, Image *d_equalizedImage, int *h_mappings) {
    int totalPixels = image->width * image->height;
    int threadsPerBlock = 512;
    int blocksPerGrid = (totalPixels + threadsPerBlock - 1) / threadsPerBlock;

    d_equalizedImage->width = image->width;
    d_equalizedImage->height = image->height;
    int *d_mappings;
    CUDA_CHECK_RETURN(hipMalloc(&d_mappings, (int) sizeof(int) * 256));
    CUDA_CHECK_RETURN(hipMemcpy(d_mappings, h_mappings, sizeof(int) * 256, hipMemcpyHostToDevice));
    CUDA_CHECK_RETURN(hipMalloc(&(d_equalizedImage->image), sizeof(unsigned char) * image->width * image->height));
    equalizeImage<< < threadsPerBlock, blocksPerGrid, 0>> > (image->image, image->width, totalPixels, d_mappings, d_equalizedImage->image);


}

// def histogramEqualization(image,histogram):
//  pdf = calculatePdf(histogram,image.shape[0]*image.shape[1])
//  cdf = calculateCdf(pdf)
//  mapLookup = np.array(cdf * 255,dtype=int)
//  shape = image.shape
//  equalized = np.zeros(shape)
//  for i in range(0,shape[0]):
//      for j in range(0,shape[1]):
//          equalized[i][j] = mapLookup[int(image[i][j])]
//  print(mapLookup)

void extractSingleColorChannel(RGBImage *rgb, Image *out, int color) {
    out->width = rgb->width;
    out->height = rgb->height;
    int totalPixels = rgb->width * rgb->height;
    //TODO: Memory leaks right here probably should fix but meh it should work well enough like this...
    switch (color) {
        case 0: // red
            out->image = rgb->image;
            break;
        case 1: // green
            out->image = rgb->image + totalPixels;

            break;
        case 2: // blue
            out->image = rgb->image + (2 * totalPixels);
            break;
        default:
            printf("invalid option\n");
            break;
    }
}

void copyHostImageToDevice(Image *host, Image *device) {
    // copy actual image data back to host from device
    CUDA_CHECK_RETURN(hipMalloc((void **) &(device->image), (int) sizeof(unsigned char) * host->width * host->height));
    CUDA_CHECK_RETURN(hipMemcpy(device->image, host->image, (int) sizeof(unsigned char) * host->width * host->height, hipMemcpyHostToDevice));
    // copy height and width to device
    device->height = host->height;
    device->width = host->width;
}

void copyDeviceImageToHost(Image *device, Image *host) {
    // copy height and width back to host
    host->height = device->height;
    host->width = device->width;
    host->image = (unsigned char *) malloc(sizeof(unsigned char) * host->height * host->width);
    // copy actual image data back to host from device
    CUDA_CHECK_RETURN(hipMemcpy(host->image, device->image, sizeof(unsigned char) * device->width * device->height, hipMemcpyDeviceToHost));
//    for (int i = 0; i < (host->height * host->width); i++) {
//        printf("%i\n", host->image[i]);
//    }

}

void copyDeviceRGBImageToHost(RGBImage *device, RGBImage *host) {
    // copy height and width back to host
    host->height = device->height;
    host->width = device->width;
    host->channels = 1;
    host->image = (unsigned char *) malloc(sizeof(unsigned char) * host->height * host->width);
    // copy actual image data back to host from device
    CUDA_CHECK_RETURN(hipMemcpy(host->image, device->image, sizeof(unsigned char) * device->width * device->height, hipMemcpyDeviceToHost));
//    for (int i = 0; i < (host->height * host->width); i++) {
//        printf("%i\n", host->image[i]);
//    }

}

void copyHostRGBImageToDevice(RGBImage *host, RGBImage *device) {
    // copy actual image data to device from host
//    unsigned char*
    CUDA_CHECK_RETURN(hipMalloc((void **) &(device->image), sizeof(unsigned char) * host->width * host->height * host->channels));
    CUDA_CHECK_RETURN(hipMemcpy(device->image, host->image, sizeof(unsigned char) * host->width * host->height * host->channels, hipMemcpyHostToDevice));
    // copy height and width to device
    device->height = host->height;
    device->width = host->width;
    device->channels = host->channels;
}

/**
 * Check the return value of the CUDA runtime API call and exit
 * the application if the call has failed.
 */
static void CheckCudaErrorAux(const char *file, unsigned line, const char *statement, hipError_t err) {
    if (err == hipSuccess)
        return;
    std::cerr<<statement<<" returned "<<hipGetErrorString(err)<<"("<<err<<") at "<<file<<":"<<line<<std::endl;
    exit(1);
}