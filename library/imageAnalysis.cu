#include "hip/hip_runtime.h"
#include "imageAnalysis.h"

#include <stdio.h>
#include <stdlib.h>
#include <iostream>

static void CheckCudaErrorAux (const char *, unsigned, const char *, hipError_t);
#define CUDA_CHECK_RETURN(value) CheckCudaErrorAux(__FILE__,__LINE__, #value, value)



/**
 * CUDA kernel that computes reciprocal values for a given vector
 */
__global__ void reciprocalKernel(float *data, unsigned vectorSize) {
    unsigned idx = blockIdx.x*blockDim.x+threadIdx.x;
    if (idx < vectorSize)
        data[idx] = 1.0f / data[idx];
}

/**
 * Host function that copies the data and launches the work on GPU
 */
float *gpuReciprocal(float *data, unsigned size)
{
    float *rc = new float[size];
    float *gpuData;

    CUDA_CHECK_RETURN(hipMalloc((void **)&gpuData, sizeof(float)*size));
    CUDA_CHECK_RETURN(hipMemcpy(gpuData, data, sizeof(float)*size, hipMemcpyHostToDevice));

    static const int BLOCK_SIZE = 256;
    const int blockCount = (size+BLOCK_SIZE-1)/BLOCK_SIZE;
    reciprocalKernel<<<blockCount, BLOCK_SIZE>>> (gpuData, size);

    CUDA_CHECK_RETURN(hipMemcpy(rc, gpuData, sizeof(float)*size, hipMemcpyDeviceToHost));
    CUDA_CHECK_RETURN(hipFree(gpuData));
    return rc;
}


void convertRGBToGrayscale(RGBImage rgb, Image gray) {

}

void copyHostImageToDevice(Image host, Image device){

}

void copyHostImageToDevice(RGBImage host, RGBImage device){

}

/**
 * Check the return value of the CUDA runtime API call and exit
 * the application if the call has failed.
 */
static void CheckCudaErrorAux (const char *file, unsigned line, const char *statement, hipError_t err)
{
    if (err == hipSuccess)
        return;
    std::cerr << statement<<" returned " << hipGetErrorString(err) << "("<<err<< ") at "<<file<<":"<<line << std::endl;
    exit (1);
}