#include "hip/hip_runtime.h"
#include "imageAnalysis.h"
#include "../library/imageAnalysis.h"

#include <stdio.h>
#include <stdlib.h>
#include <sys/ioctl.h>
#include <math.h>
#include <string.h>
# include <assert.h>

#include <vector>
#include <iostream>
#include <fstream>
#include <regex>

#include <dirent.h>
#include <opencv2/opencv.hpp>
#include <json/json.h>

using namespace std;
using namespace cv;
#define LOGLEVEL 5
//#define DEBUG_GRAYSCALE true
#define DEBUG_GRAYSCALE false
//#define DEBUG_HIST true
#define DEBUG_HIST false
//#define DEBUG_EQUALIZED true
#define DEBUG_EQUALIZED false
//#define DEBUG_LINFILTER true
#define DEBUG_LINFILTER false
//#define DEBUG_MEDFILTER true
#define DEBUG_MEDFILTER false
// I don't write very memory efficient c code and tend to introduce some memory leakage but oh well today isn't the day I figure it out...


vector<int> compression_params;

void convertMatToRGBImage(Mat mat, RGBImage *output) {
    Mat bgr[3];
    split(mat, bgr);
    output->channels = mat.channels();
    output->height = mat.rows;
    output->width = mat.cols;
    output->image = ((unsigned char *) malloc(sizeof(unsigned char) * mat.total() * output->channels));
    int numPixels = mat.total();
    // swapping into rgb format here instead of the bgr the OpenCV Mat is in
    int row = 0, col = 0;
    for (int i = 0; i < numPixels; i++) {
        row = i / numPixels;
        col = i - ((i / numPixels) * numPixels);
        output->image[i] = bgr[2].at<uchar>(row, col);
        output->image[i + numPixels] = bgr[1].at<uchar>(row, col);
        output->image[i + numPixels * 2] = bgr[0].at<uchar>(row, col);
    }
}


void convertRGBImageToMat(RGBImage *image, Mat *output) {
    // adapted from https://stackoverflow.com/a/43190162
    int numPixels = image->height * image->width;
    Mat channelR(image->height, image->width, CV_8UC1, image->image);
    Mat channelG(image->height, image->width, CV_8UC1, image->image + numPixels);
    Mat channelB(image->height, image->width, CV_8UC1, image->image + 2 * numPixels);
    std::vector <Mat> channels{channelB, channelG, channelR};

    merge(channels, *output);
}

void convertImageToMat(Image *image, Mat *mat) {
    Mat output(image->height, image->width, CV_8UC1, image->image);
    *mat = output;
}

void readInKernel(Json::Value kernel, float *k, int numValues) {
    const Json::Value &k_vals = kernel["values"];
    assert(numValues == k_vals.size());
    for (int i = 0; i < numValues; i++) {
        k[i] = k_vals[i].asFloat();
    }
}

void readInKernel(Json::Value kernel, int *k, int numValues) {
    const Json::Value &k_vals = kernel["values"];
    assert(numValues == k_vals.size());
    for (int i = 0; i < numValues; i++) {
        k[i] = k_vals[i].asInt();
    }
}

vector <string> getFileNames(string input_image_folder, regex filter) {
    // adapted from this https://stackoverflow.com/a/612176
    vector <string> files;
    DIR *dir;
    struct dirent *ent;
    if ((dir = opendir(input_image_folder.c_str())) != NULL) {
        /* print all the files and directories within directory */
        while ((ent = readdir(dir)) != NULL) {
            if (strncmp(ent->d_name, ".", 1)) {
                if (regex_search(ent->d_name, filter))
                    files.push_back(ent->d_name);
//                printf("%s\n", ent->d_name);
            }
        }
        closedir(dir);
    } else {
        perror("");
    }

    std::sort(files.begin(), files.end());
    return files;

}

void saveImage(string output_image_folder, Image *d_image, Image *h_image, Mat *outputMat, string type, string fileName) {
    copyDeviceImageToHost(d_image, h_image);
    convertImageToMat(h_image, outputMat);
    string outPath = output_image_folder + "/" + fileName;
    if (type.length() > 0) {
        outPath = output_image_folder + "/" + type + "-" + fileName;
    }

    if (LOGLEVEL >= 5)
        printf("writing to %s\n", outPath.c_str());
    imwrite(outPath, *outputMat);//, compression_params);
}

void executeOperations(Json::Value json, string input_image_folder, string output_image_folder, bool saveFinalImages, bool saveIntermediateImages, string extract_channel, regex fileFilter,
                       bool calcMSQEConfig) {
    vector <string> files = getFileNames(input_image_folder, fileFilter);
    const Json::Value &operations = json["operations"];
    int numOperations = operations.size();
    string curFilePath;
    int k_width;
    int k_height;
    float *kern;
    int *medKern;
    Mat mat;
    int *h_histogram = nullptr;
    int *d_histogram = nullptr;
    int h_mappings[256];
    hipHostMalloc(&h_histogram, sizeof(int) * 256);
    RGBImage *h_rgbImage = new RGBImage;
    RGBImage *d_rgbImage = new RGBImage;
    Image *d_image = new Image;
    Image *d_tempImage = new Image;
    Image *h_image = new Image;
    Mat *outputMat = new Mat;
//    types of cells: cyl inter let mod para super svar


//    hipStream_t *stream = new hipStream_t;
    hipEvent_t operationStart, operationStop, batchStart, batchStop;
    hipEventCreate(&operationStart);
    hipEventCreate(&operationStop);
    hipEventCreate(&batchStart);
    hipEventCreate(&batchStop);
    float milliseconds = 0;
    float totalBatchTime = 0,
            totalGrayscaleTime = 0,
            totalSingleChannelConvertTime = 0,
            totalGaussianNoiseTime = 0,
            totalSaltAndPepperNoiseTime = 0,
            totalHistEqualizationTime = 0,
            totalQuantizationTime = 0,
            totalLinearFilterTime = 0,
            totalAverageFilterTime = 0,
            totalMedianFilterTime = 0;
    float totalMSQE = 0;
    float numImages = files.size();
    Image *h_equalizedImage = new Image;
    bool randomnessSet = false;
    hipEventRecord(batchStart);

    for (int k = 0; k < files.size(); k++) { // iterate through all the images in the folder
        curFilePath = files[k];
        if (LOGLEVEL >= 4)
            printf("Working on image %s\n", curFilePath.c_str());
        try {
            mat = imread(input_image_folder + "/" + curFilePath, CV_LOAD_IMAGE_COLOR);
            convertMatToRGBImage(mat, h_rgbImage);
            // convert image to a single color spectrum
            if (extract_channel == "grey") {
                hipEventRecord(operationStart);

                copyHostRGBImageToDevice(h_rgbImage, d_rgbImage);
                convertRGBToGrayscale(d_rgbImage, d_image, 0);

                hipEventRecord(operationStop);
                hipEventSynchronize(operationStop);
                hipEventElapsedTime(&milliseconds, operationStart, operationStop);
                totalGrayscaleTime += milliseconds;
            } else if (extract_channel == "red") {
                hipEventRecord(operationStart);

                extractSingleColorChannel(h_rgbImage, h_image, 0);
                copyHostImageToDevice(h_image, d_image);

                hipEventRecord(operationStop);
                hipEventSynchronize(operationStop);
                hipEventElapsedTime(&milliseconds, operationStart, operationStop);
                totalSingleChannelConvertTime += milliseconds;
            } else if (extract_channel == "green") {
                hipEventRecord(operationStart);

                extractSingleColorChannel(h_rgbImage, h_image, 1);
                copyHostImageToDevice(h_image, d_image);

                hipEventRecord(operationStop);
                hipEventSynchronize(operationStop);
                hipEventElapsedTime(&milliseconds, operationStart, operationStop);
                totalSingleChannelConvertTime += milliseconds;
            } else if (extract_channel == "blue") {
                hipEventRecord(operationStart);

                extractSingleColorChannel(h_rgbImage, h_image, 2);
                copyHostImageToDevice(h_image, d_image);

                hipEventRecord(operationStop);
                hipEventSynchronize(operationStop);
                hipEventElapsedTime(&milliseconds, operationStart, operationStop);
                totalSingleChannelConvertTime += milliseconds;
            } else {
                printf("Unsupported color option: %s\n", extract_channel.c_str());
                exit(-10);
            }
            if (saveIntermediateImages) {
                saveImage(output_image_folder, d_image, h_image, outputMat, extract_channel, curFilePath);
            }
            if (!randomnessSet) {
                setupRandomness(d_image);
                randomnessSet = true;
            }

            for (int i = 0; i < numOperations; i++) { // perform the operations on each image
                bool supported = true;

                string type = operations[i]["type"].asString();
                if (type == "linear-filter") {
                    Json::Value kernel = operations[i]["kernel"];
                    k_width = kernel["width"].asInt();
                    k_height = kernel["height"].asInt();
                    kern = (float *) malloc(sizeof(float) * k_width * k_height);
                    readInKernel(kernel, kern, k_width * k_height);
                    hipEventRecord(operationStart);
                    linearFilter(d_image, d_tempImage, kern, k_width, k_height);
                    CUDA_CHECK_RETURN(hipFree(d_image->image));
                    d_image->image = d_tempImage->image;
                    hipEventRecord(operationStop);
                    hipEventSynchronize(operationStop);
                    hipEventElapsedTime(&milliseconds, operationStart, operationStop);
                    totalLinearFilterTime += milliseconds;
                    free(kern);
                } else if (type == "average-filter") {
                    Json::Value kernel = operations[i]["kernel"];
                    k_width = kernel["width"].asInt();
                    k_height = kernel["height"].asInt();
                    kern = (float *) malloc(sizeof(float) * k_width * k_height);
                    readInKernel(kernel, kern, k_width * k_height);
                    hipEventRecord(operationStart);
                    averageFilter(d_image, d_tempImage, kern, k_width, k_height);
                    CUDA_CHECK_RETURN(hipFree(d_image->image));
                    d_image->image = d_tempImage->image;
                    hipEventRecord(operationStop);
                    hipEventSynchronize(operationStop);
                    hipEventElapsedTime(&milliseconds, operationStart, operationStop);
                    totalAverageFilterTime += milliseconds;
                    free(kern);
                } else if (type == "median-filter") {
                    Json::Value kernel = operations[i]["kernel"];
                    k_width = kernel["width"].asInt();
                    k_height = kernel["height"].asInt();
                    medKern = (int *) malloc(sizeof(int) * k_width * k_height);
                    readInKernel(kernel, medKern, k_width * k_height);
                    hipEventRecord(operationStart);
                    medianFilter(d_image, d_tempImage, medKern, k_width, k_height);
                    CUDA_CHECK_RETURN(hipFree(d_image->image));
                    d_image->image = d_tempImage->image;
                    hipEventRecord(operationStop);
                    hipEventSynchronize(operationStop);
                    hipEventElapsedTime(&milliseconds, operationStart, operationStop);
                    totalMedianFilterTime += milliseconds;
                    free(medKern);
                } else if (type == "gaussian-noise") {
                    hipEventRecord(operationStart);
                    float stdDev = operations[i]["std_dev"].asFloat();
                    float mean = operations[i]["mean"].asFloat();
                    addGaussianNoise(d_image, d_tempImage, mean, stdDev);
                    CUDA_CHECK_RETURN(hipFree(d_image->image));
                    d_image->image = d_tempImage->image;
                    hipEventRecord(operationStop);
                    hipEventSynchronize(operationStop);
                    hipEventElapsedTime(&milliseconds, operationStart, operationStop);
                    totalGaussianNoiseTime += milliseconds;
                } else if (type == "salt-and-pepper") {
                    int level = operations[i]["intensity"].asInt();
                    hipEventRecord(operationStart);
                    saltAndPepperNoise(d_image, d_tempImage, level);
                    CUDA_CHECK_RETURN(hipFree(d_image->image));
                    d_image->image = d_tempImage->image;
                    hipEventRecord(operationStop);
                    hipEventSynchronize(operationStop);
                    hipEventElapsedTime(&milliseconds, operationStart, operationStop);
                    totalSaltAndPepperNoiseTime += milliseconds;
                } else if (type == "histogram-equalization") {
                    hipEventRecord(operationStart);
                    calculateHistogram(d_image, h_histogram, d_histogram);
                    equalizeHistogram(h_histogram, h_mappings, d_image->height * d_image->width);
                    equalizeImageWithHist(d_image, d_tempImage, h_mappings);
                    CUDA_CHECK_RETURN(hipFree(d_image->image));
                    d_image->image = d_tempImage->image;
                    hipEventRecord(operationStop);
                    hipEventSynchronize(operationStop);
                    hipEventElapsedTime(&milliseconds, operationStart, operationStop);
                    totalHistEqualizationTime += milliseconds;
                } else if (type == "quantization") {
                    const Json::Value &levelsJson = operations[i]["levels"];
                    int numLevels = levelsJson.size();
                    int *levels = (int *) malloc(sizeof(int) * 3 * numLevels);
                    for (int v = 0; v < numLevels; v++) {
                        Json::Value levelJson = levelsJson[v];
                        levels[v * 3] = levelJson["min"].asInt();
                        levels[v * 3 + 1] = levelJson["max"].asInt();
                        levels[v * 3 + 2] = levelJson["val"].asInt();
                    }
                    hipEventRecord(operationStart);
                    imageQuantization(d_image, d_tempImage, levels, numLevels);
                    if (calcMSQEConfig) {
                        int MSQE = calcMSQE(d_image, d_tempImage);
                        printf("MSQE of imageQuantization is %i\n", MSQE);
                        totalMSQE += MSQE;
                    }
                    d_image->image = d_tempImage->image;
                    free(levels);
                    hipEventRecord(operationStop);
                    hipEventSynchronize(operationStop);
                    hipEventElapsedTime(&milliseconds, operationStart, operationStop);
                    totalQuantizationTime += milliseconds;
                } else {
                    printf("Unsupported Operation\n");
                    supported = false;
                }
                // copy images back to host and save intermediates if configured to do so...
                if (saveIntermediateImages && supported) {
                    saveImage(output_image_folder, d_image, h_image, outputMat, type, curFilePath);
                }
                supported = true;
            }
            // copy device image back to host and save it if configured to do so...
            if (saveFinalImages) {
                saveImage(output_image_folder, d_image, h_image, outputMat, "", curFilePath);
            }
            hipEventRecord(batchStop);
            hipEventSynchronize(batchStop);
            hipEventElapsedTime(&milliseconds, batchStart, batchStop);
            totalBatchTime += milliseconds;
        } catch (const std::exception &e) {
            printf("Some sort of issue processing image %s\n", curFilePath.c_str());
            hipError_t error = hipGetLastError();
            if (error != hipSuccess) {
                printf("CUDA error: %s\n", hipGetErrorString(error));
            }
            numImages--;
            continue;
        }
    }
    printf("\n\nTotal time spent on the entire batch: %0.4f ms average of %0.4f ms for each image\n", totalBatchTime, totalBatchTime / numImages);
    if (extract_channel == "grey") {
        printf("Total time spent converting to grayscale: %0.4f ms average of: %0.4f ms per image\n", totalGrayscaleTime, totalGrayscaleTime / numImages);
    } else {
        printf("Total time spent converting to a single channel: %0.4f ms average of: %0.4f ms per image\n", totalSingleChannelConvertTime, totalSingleChannelConvertTime / numImages);
    }
    printf("Total time spent performing histogram equalization: %0.4f ms average of: %0.4f ms per image\n", totalHistEqualizationTime, totalHistEqualizationTime / numImages);
    printf("Total time spent adding gaussian noise: %0.4f ms average of: %0.4f ms per image\n", totalGaussianNoiseTime, totalGaussianNoiseTime / numImages);
    printf("Total time spent adding salt and pepper noise: %0.4f ms average of: %0.4f ms per image\n", totalSaltAndPepperNoiseTime, totalSaltAndPepperNoiseTime / numImages);
    if (calcMSQEConfig) {
        printf("Total time spent performing image quantization: %0.4f ms average of: %0.4f ms per image with an average MSQE of %0.4f\n", totalQuantizationTime, totalQuantizationTime / numImages,
               totalMSQE / numImages);
    } else {
        printf("Total time spent performing image quantization: %0.4f ms average of: %0.4f ms per image\n", totalQuantizationTime, totalQuantizationTime / numImages);
    }
    printf("Total time spent linear filtering image: %0.4f ms average of: %0.4f ms per image\n", totalLinearFilterTime, totalLinearFilterTime / numImages);
    printf("Total time spent average filtering image: %0.4f ms average of: %0.4f ms per image\n", totalAverageFilterTime, totalAverageFilterTime / numImages);
    printf("Total time spent median filtering image: %0.4f ms average of: %0.4f ms per image\n", totalMedianFilterTime, totalMedianFilterTime / numImages);

}


int main(int argc, char *argv[]) {
    compression_params.push_back(CV_IMWRITE_PNG_COMPRESSION);
    compression_params.push_back(9);
    Json::Value json;
    if (argc < 2) {
        printf("please pass a config file path as the argument\n");
        exit(-1);
    }
    std::ifstream config(argv[1], std::ifstream::binary);
    config>>json;
    string input_image_folder = json["image_folder"].asString();
    string output_image_folder = json["output_dir"].asString();
    string extract_channel = json["extract_channel"].asString();
    regex fileFilter = regex(json["input_image_filter"].asString());
    bool saveFinalImages = json["saveFinalImages"].asBool();
    bool saveIntermediateImages = json["saveIntermediateImages"].asBool();
    bool calcMSQEConfig = json["calc_MSQE"].asBool();
    printf("Input: %s\nOutput: %s\nSaving intermediates: %s\nSaving Finals: %s\n",
           input_image_folder.c_str(),
           output_image_folder.c_str(),
           saveIntermediateImages ? "true" : "false",
           saveFinalImages ? "true" : "false");
    executeOperations(json, input_image_folder, output_image_folder, saveFinalImages, saveIntermediateImages, extract_channel, fileFilter, calcMSQEConfig);


    return 0;
}

